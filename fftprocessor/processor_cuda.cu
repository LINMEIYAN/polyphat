#include "hip/hip_runtime.h"

#include <memory.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "settings.h"


extern "C" {
#include "processor_cuda.h"
}

#include <stdio.h>
#include "math.h"

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define CHK( err ) (HandleError( err, __FILE__, __LINE__ ))


uint8_t* alloc_sample_buffer() {
  uint8_t *p;
  /* size of uint8_t is one, explicityly */
  CHK(hipHostAlloc(&p, BUFFER_SIZE, hipHostMallocDefault));
 return p;
}


void print_timing (hipEvent_t* start, hipEvent_t* stop, char* what) {
  float gpu_time;
  CHK(hipEventElapsedTime(&gpu_time, *start, *stop));
  printf ("Timing %s : %fms \n",what, gpu_time);
}


#define FLOATIZE_X 8
/**
 * CUDA Kernel byte->float
 *
 */
__global__ void floatize(uint8_t* sample,hipfftReal* fsample)  {
    int i = FLOATIZE_X*(blockDim.x * blockIdx.x + threadIdx.x);
    for (int j=0; j<FLOATIZE_X; j++) fsample[i+j]=float(sample[i+j]-128);
}




void cuda_test(uint8_t *buf) {

  // cuda buffer and float buffer
  uint8_t *cbuf;
  CHK(hipMalloc(&cbuf,BUFFER_SIZE));
  hipfftReal *cfbuf;
  CHK(hipMalloc(&cfbuf,BUFFER_SIZE*sizeof(hipfftReal)));
  hipfftComplex *ffts;
  CHK(hipMalloc(&ffts,TRANSFORM_SIZE*NUM_FFT*sizeof(hipfftComplex)));

  hipfftHandle plan;
  int oembed=TRANSFORM_SIZE*NUM_FFT+1;
  int fftsize=FFT_SIZE;
  int status=hipfftPlanMany(&plan, 1, &fftsize, NULL, 0, 0, 
        NULL, TRANSFORM_SIZE,1, HIPFFT_R2C, NUM_FFT);
  if (status!=HIPFFT_SUCCESS) {
       printf ("Plan failed:");
       if (status==HIPFFT_ALLOC_FAILED) printf("HIPFFT_ALLOC_FAILED");
       if (status==HIPFFT_INVALID_VALUE) printf ("HIPFFT_INVALID_VALUE");
       if (status==HIPFFT_INTERNAL_ERROR) printf ("HIPFFT_INTERNAL_ERROR");
       if (status==HIPFFT_SETUP_FAILED) printf ("HIPFFT_SETUP_FAILED");
       if (status==HIPFFT_INVALID_SIZE) printf ("HIPFFT_INVALID_SIZE");
       printf("\n");
       exit(1);
  }

  hipEvent_t tstart, tcpy,tfloatize,tfft;
  CHK(hipEventCreate(&tstart));
  CHK(hipEventCreate(&tcpy));
  CHK(hipEventCreate(&tfloatize));
  CHK(hipEventCreate(&tfft));

  hipEventRecord(tstart, 0);
  // copy to device
  CHK(hipMemcpy(cbuf,buf, BUFFER_SIZE, hipMemcpyHostToDevice));

  hipEventRecord(tcpy, 0);
  
  // floatize
  int threadsPerBlock = 1024;
  int blocksPerGrid = BUFFER_SIZE / threadsPerBlock/FLOATIZE_X;
  floatize<<<blocksPerGrid, threadsPerBlock>>>(cbuf,cfbuf);
  hipEventRecord(tfloatize, 0);
  CHK(hipGetLastError());
  
  status=hipfftExecR2C(plan, cfbuf, ffts);
  hipEventRecord(tfft, 0);
  if (status!=HIPFFT_SUCCESS) {
     printf("CUFFT FAILED\n");
     exit(1);
  }    

  hipDeviceSynchronize();
  print_timing(&tstart,&tcpy,"MEM CPY");
  print_timing(&tcpy,&tfloatize,"FLOATIZE");
  print_timing(&tfloatize,&tfft,"FFT");

}


void ztest() {
  uint8_t *cbuf;
  CHK(hipMalloc(&cbuf,BUFFER_SIZE));
  // floatize
  int threadsPerBlock = 1024;
  int blocksPerGrid = 32768;
  int Nth=
  printf ("%i %i",threadsPerBlock, blocksPerGrid);
  //  floatize<<<blocksPerGrid, threadsPerBlock>>>(cbuf);
  CHK(hipGetLastError());

}
