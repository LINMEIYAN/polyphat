#include "hip/hip_runtime.h"

#include <memory.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "settings.h"

extern "C" {
#include "processor_cuda.h"
}

#include <stdio.h>
#include "math.h"

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define CHK( err ) (HandleError( err, __FILE__, __LINE__ ))


uint8_t* alloc_sample_buffer() {
  uint8_t *p;
  /* size of uint8_t is one, explicityly */
  CHK(hipHostAlloc(&p, BUFFER_SIZE, hipHostMallocDefault));
 return p;
}


void print_timing (hipEvent_t* start, hipEvent_t* stop, char* what) {
  float gpu_time;
  CHK(hipEventElapsedTime(&gpu_time, *start, *stop));
  printf ("Timing %s : %fms \n",what, gpu_time);
}


#define FLOATIZE_X 8
/**
 * CUDA Kernel byte->float
 *
 */
__global__ void floatize(uint8_t* sample,float* fsample)  {
    int i = FLOATIZE_X*(blockDim.x * blockIdx.x + threadIdx.x);
    for (int j=0; j<FLOATIZE_X; j++) fsample[i+j]=float(sample[i+j]-128);
}




void cuda_test(uint8_t *buf) {

  // cuda buffer and float buffer
  uint8_t *cbuf;
  CHK(hipMalloc(&cbuf,BUFFER_SIZE));
  float *cfbuf;
  CHK(hipMalloc(&cfbuf,BUFFER_SIZE*sizeof(float)));

  hipEvent_t tstart, tcpy,tfloatize;
  CHK(hipEventCreate(&tstart));
  CHK(hipEventCreate(&tcpy));
  CHK(hipEventCreate(&tfloatize));

  hipEventRecord(tstart, 0);
  // copy to device
  CHK(hipMemcpy(cbuf,buf, BUFFER_SIZE, hipMemcpyHostToDevice));

  hipEventRecord(tcpy, 0);
  
  // floatize
  int threadsPerBlock = 1024;
  int blocksPerGrid = BUFFER_SIZE / threadsPerBlock/FLOATIZE_X;
  floatize<<<blocksPerGrid, threadsPerBlock>>>(cbuf,cfbuf);
  hipEventRecord(tfloatize, 0);
  CHK(hipGetLastError());
  
  //



  hipDeviceSynchronize();
  print_timing(&tstart,&tcpy,"MEM CPY");
  print_timing(&tcpy,&tfloatize,"FLOATIZE");

}


void ztest() {
  uint8_t *cbuf;
  CHK(hipMalloc(&cbuf,BUFFER_SIZE));
  // floatize
  int threadsPerBlock = 1024;
  int blocksPerGrid = 32768;
  int Nth=
  printf ("%i %i",threadsPerBlock, blocksPerGrid);
  //  floatize<<<blocksPerGrid, threadsPerBlock>>>(cbuf);
  CHK(hipGetLastError());

}
