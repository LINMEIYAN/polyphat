#include "hip/hip_runtime.h"

#include <memory.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "settings.h"

extern "C" {
#include "processor_cuda.h"
}

#include <stdio.h>
#include "math.h"

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define CHK( err ) (HandleError( err, __FILE__, __LINE__ ))


uint8_t* alloc_sample_buffer() {
  uint8_t *p;
  /* size of uint8_t is one, explicityly */
  CHK(hipHostAlloc(&p, BUFFER_SIZE, hipHostMallocDefault));
 return p;
}



/**
 * CUDA Kernel byte->float
 *
 */
__global__ void floatize(uint8_t *sample, float* fsample)  {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<BUFFER_SIZE) fsample[i]=float(sample[i]-128);
}


void print_timing (hipEvent_t start, hipEvent_t stop, char* what) {
  float gpu_time;
  hipEventElapsedTime(&gpu_time, start, stop);
  printf ("Timing %s : %fms ",what, gpu_time);
}



void cuda_test(uint8_t *buf) {

  // cuda buffer and float buffer
  uint8_t *cbuf;
  CHK(hipMalloc(&cbuf,BUFFER_SIZE));
  float *cfbuf;
  CHK(hipMalloc(&cfbuf,BUFFER_SIZE*sizeof(float)));

  hipEvent_t tstart, tcpy,tfloatize;
  CHK(hipEventCreate(&tstart));
  CHK(hipEventCreate(&tcpy));
  CHK(hipEventCreate(&tfloatize));

  hipEventRecord(tstart, 0);
  // copy to device
  hipMemcpy(cbuf,buf, BUFFER_SIZE, hipMemcpyHostToDevice);
  // floatize
  hipEventRecord(tcpy, 0);

  int threadsPerBlock = 256;
  int blocksPerGrid = BUFFER_SIZE / threadsPerBlock;
  
  floatize<<<blocksPerGrid, threadsPerBlock>>>(cbuf, cfbuf);
  CHK(hipGetLastError());
  hipEventRecord(tfloatize, 0);


}

