#include "hip/hip_runtime.h"
/***********************************
***********************************
CUDA PART
***********************************
**********************************/

#include "gpucard.h"

#include <memory.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include <stdio.h>
#include "math.h"

#define FLOATIZE_X 8


static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
      printf( "CUDA fail: %s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit(1);
    }
}
#define CHK( err ) (HandleError( err, __FILE__, __LINE__ ))

void gpuCardInit (GPUCARD *gc, SETTINGS *set) {
  printf ("\n\nInitilizing GPU\n");
  printf ("=================\n");
  printf ("Allocating GPU buffers\n");
  int Nb=set->buf_mult;
  gc->cbuf=(void**)malloc(Nb*sizeof(void*));
  gc->cfbuf=(void**)malloc(Nb*sizeof(void*));
  gc->cfft=(void**)malloc(Nb*sizeof(void*));
  int nchan=gc->nchan=1+(set->channel_mask==3);
  if ((nchan==2) and (FLOATIZE_X%2==1)) {
    printf ("Need FLOATIZE_X even for two channels\n");
    exit(1);
  }
  gc->fftsize=set->fft_size;
  uint32_t bufsize=gc->bufsize=set->fft_size*nchan;
  uint32_t transform_size=(set->fft_size/2+1)*nchan;
  for (int i=0;i<Nb;i++) {
    uint8_t** cbuf=(uint8_t**)&(gc->cbuf[i]);
    CHK(hipMalloc(cbuf,bufsize));
    hipfftReal** cfbuf=(hipfftReal**)&(gc->cfbuf[i]);
    CHK(hipMalloc(cfbuf, bufsize*sizeof(hipfftReal)));
    hipfftComplex** ffts=(hipfftComplex**)&(gc->cfft[i]);
    CHK(hipMalloc(ffts,transform_size*sizeof(hipfftComplex)));
  }


  printf ("Setting up CUFFT");
  int status=hipfftPlanMany(&gc->plan, 1, (int*)&(set->fft_size), NULL, 0, 0, 
        NULL, transform_size,1, HIPFFT_R2C, nchan);

  if (status!=HIPFFT_SUCCESS) {
       printf ("Plan failed:");
       if (status==HIPFFT_ALLOC_FAILED) printf("HIPFFT_ALLOC_FAILED");
       if (status==HIPFFT_INVALID_VALUE) printf ("HIPFFT_INVALID_VALUE");
       if (status==HIPFFT_INTERNAL_ERROR) printf ("HIPFFT_INTERNAL_ERROR");
       if (status==HIPFFT_SETUP_FAILED) printf ("HIPFFT_SETUP_FAILED");
       if (status==HIPFFT_INVALID_SIZE) printf ("HIPFFT_INVALID_SIZE");
       printf("\n");
       exit(1);
  }
  printf ("Setting up CUDA streams & events\n");
  gc->nstreams=set->cuda_streams;
  if (gc->nstreams<1) {
    printf ("Cannot relly work with less than one stream.\n");
    exit(1);
  }
  gc->streams=malloc(gc->nstreams*sizeof(hipStream_t));

  gc->eStart=malloc(gc->nstreams*sizeof(hipEvent_t));
  gc->eDoneCopy=malloc(gc->nstreams*sizeof(hipEvent_t));
  gc->eDoneFloatize=malloc(gc->nstreams*sizeof(hipEvent_t));
  gc->eDoneFFT=malloc(gc->nstreams*sizeof(hipEvent_t));
  gc->eDonePost=malloc(gc->nstreams*sizeof(hipEvent_t));
  hipEvent_t* eStart=(hipEvent_t*)(gc->eStart);
  hipEvent_t* eDoneCopy=(hipEvent_t*)(gc->eDoneCopy);
  hipEvent_t* eDoneFloatize=(hipEvent_t*)(gc->eDoneFloatize);
  hipEvent_t* eDoneFFT=(hipEvent_t*)(gc->eDoneFFT);
  hipEvent_t* eDonePost=(hipEvent_t*)(gc->eDonePost);

  for (int i=0;i<gc->nstreams;i++) {
    CHK(hipEventCreate(&eStart[i]));
    CHK(hipEventCreate(&eDoneCopy[i]));
    CHK(hipEventCreate(&eDoneFloatize[i]));
    CHK(hipEventCreate(&eDoneFFT[i]));
    CHK(hipEventCreate(&eDonePost[i]));
  }
  gc->fstream=gc->bstream=gc->active_streams=0;
}




/**
 * CUDA Kernel byte->float, 1 channel version
 *
 */
__global__ void floatize_1chan(uint8_t* sample, hipfftReal* fsample)  {
    int i = FLOATIZE_X*(blockDim.x * blockIdx.x + threadIdx.x);
    for (int j=0; j<FLOATIZE_X; j++) fsample[i+j]=float(sample[i+j]);
}

__global__ void floatize_2chan(uint8_t* sample, hipfftReal* fsample1, hipfftReal* fsample2)  {
    int i = FLOATIZE_X*(blockDim.x * blockIdx.x + threadIdx.x);
    for (int j=0; j<FLOATIZE_X/2; j++) {
      fsample1[i+j]=float(sample[i+2*j]);
      fsample2[i+j]=float(sample[i+2*j+1]);
    }
}


bool gpuProcessBuffer(GPUCARD *gc, int8_t *buf) {

  // pointers and vars
  uint8_t** cbuf=(uint8_t**)(gc->cbuf);
  hipfftReal** cfbuf=(hipfftReal**)(gc->cfbuf);
  hipfftComplex** cfft=(hipfftComplex**)(gc->cfft);

  hipEvent_t* eStart=(hipEvent_t*)(gc->eStart);
  hipEvent_t* eDoneCopy=(hipEvent_t*)(gc->eDoneCopy);
  hipEvent_t* eDoneFloatize=(hipEvent_t*)(gc->eDoneFloatize);
  hipEvent_t* eDoneFFT=(hipEvent_t*)(gc->eDoneFFT);
  hipEvent_t* eDonePost=(hipEvent_t*)(gc->eDonePost);
  hipStream_t* streams=(hipStream_t*)gc->streams;

  // first check if there are buffers to store
  while (gc->active_streams>0) {
    // process done streams
    // IMPLEMENT
  }
  // add a new stream
  gc->active_streams++;
  int csi=gc->bstream = (++gc->bstream)%(gc->nstreams);
  hipStream_t cs= streams[gc->bstream];
  hipEventRecord(eStart[csi], cs);
  CHK(hipMemcpyAsync(cbuf[csi], buf, gc->bufsize , hipMemcpyHostToDevice,cs));
  hipEventRecord(eDoneCopy[csi], cs);
  int threadsPerBlock = gc->threads;
  int blocksPerGrid = gc->bufsize / threadsPerBlock/FLOATIZE_X;
  if (gc->nchan==1) 
    floatize_1chan<<<blocksPerGrid, threadsPerBlock, 0, cs>>>(cbuf[csi],cfbuf[csi]);
  else 
    floatize_2chan<<<blocksPerGrid, threadsPerBlock, 0, cs>>>(cbuf[csi],cfbuf[csi],&(cfbuf[csi][gc->fftsize]));
  hipEventRecord(eDoneFloatize[csi], cs);
  int status=hipfftExecR2C(gc->plan, cfbuf[csi], cfft[csi]);
  
  hipEventRecord(eDoneFFT[csi], cs);
  hipEventRecord(eDonePost[csi], cs);




  
  return true;
}
